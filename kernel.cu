
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
using namespace std;
#define N 3 //rowsize
#define M 4 // columnsize
const int blockNUM = 4;
const int threadNUM =3;
void mxv(const int rowsize,const int columnsize,
         const float*matrix,const float*v,float*r)
         {
             for(int i=0;i<rowsize;i++)
             {
                 float re=0.0f;
                 for(int j=0;j<columnsize;j++)
                    re+=(matrix[i*columnsize+j]*v[j]);
                 r[i]=re;
             }
			 cout <<"CPU:";
             for(int i=0;i<rowsize;i++)
                cout << r[i]<< " ";
             cout <<endl;
         }

static void __global__ mxvNaive(int rowSize, int columnSize, int columnPitch,
const float *d_matrix, const float *d_vec, float *d_r)
  {
      int id = threadIdx.x+blockIdx.x*blockDim.x;
      if(id<rowSize)
      {
          float temp=0;
          for(int i=0;i<columnSize;i++)
          {
              temp+=d_matrix[id*columnPitch+i]*d_vec[i];
          }
          d_r[id]=temp;
      }
}
int main()
{
    float *matrix=(float*)malloc(N*M*sizeof(float));
	float *vec=(float*)malloc(M*sizeof(float));
	float *r =(float*)malloc(N*sizeof(float));
	float *dev_matrix,*dev_vec,*dev_r;
	hipMalloc((void**)&dev_vec,M*sizeof(float));
	hipMalloc((void**)&dev_matrix,M*N*sizeof(float));
	hipMalloc((void**)&dev_r,N*sizeof(float));
	for(int i=1;i<=N*M;i++)
		matrix[i-1]=i;
		for(int i=0;i<M;i++)
            vec[i]=i+1;
	hipMemcpy(dev_matrix,matrix,M*N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_vec,vec,M*sizeof(float),hipMemcpyHostToDevice);
	mxvNaive<<<blockNUM,threadNUM>>> (N,M,M,dev_matrix,dev_vec,dev_r);
	hipMemcpy(r,dev_r,N*sizeof(float),hipMemcpyDeviceToHost);
	cout << "GPU:";
	for(int i=0;i<N;i++)
		cout <<r[i]<<" ";
	cout <<endl;
	mxv(N,M,matrix,vec,r);
    return 0;
}
